
#include <hip/hip_runtime.h>
#include <stdio.h> 
#define N 1024

__global__ void vectorAdd(int *a, int *b, int *c){
	int i= threadIdx.x;
	*(c+i) = *(a+i) + *(b+i);
}

int main(void)
{
	int *a,*b,*c;
	hipMallocManaged(&a,N*sizeof(int));
	hipMallocManaged(&b,N*sizeof(int));
	hipMallocManaged(&c,N*sizeof(int));
	for(int i = 0; i < N; ++i){
		*(a+i) = 2*i;
		*(b+i) = i;
		*(c+i) = 0;
	}

	vectorAdd<<<1,N>>>(a,b,c);

	hipDeviceSynchronize();
	
	for(int i = 0; i < 10 ; ++i){
		printf("c[%d] = %d\n",i,c[i]);
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}
