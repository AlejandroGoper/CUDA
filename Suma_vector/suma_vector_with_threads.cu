
#include <hip/hip_runtime.h>
#include <iostream>
//#include <cstdlib>

using namespace std;

/*
    Definición del Kernel 
*/
__global__ void sumar(int *a, int *b, int *r){
    int i = threadIdx.x;
    *(r+i) = *(a+i) + *(b+i);
}

void random_ints(int *r, int n){
    
    for(int i = 0; i<n ; i++){
        *(r+i) = rand()%5000;
    }
}

int main(int argc, char** argv){
    int N = 10;
    int *a, *b, *c; // Copias del Host
    int *d_a,*d_b,*d_c; // Copias del Device

    //Reservo memoria en la CPU
    a = new int[N];
    b = new int[N];
    c = new int[N];
     
    //Lleno a y b de valores aleatorios.
    random_ints(a,N);
    random_ints(b,N);

    //Imprimiendo 
    for(int i = 0; i<N; i++){
        cout << *(a+i) << "\t" << *(b+i) << endl;
    }

    int size = N*sizeof(int);

    //Reservo memoria en la GPU
    hipMalloc((void **)&d_a,size);
    hipMalloc((void **)&d_b,size);
    hipMalloc((void **)&d_c,size);

    //Copio valores del Host al Device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    //Lanzo el kernel

    sumar<<<1,N>>>(d_a,d_b,d_c);

    //Copio el resultado de Device a Host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    cout << endl;
    for(int i = 0; i < N; i++){
        cout << *(c+i) << endl;
    } 

    //Limpio variables de CPU
    delete [] a;
    delete [] b;
    delete [] c;
 
    // Lipio variables de GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}