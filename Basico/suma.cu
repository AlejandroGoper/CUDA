
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

/*
    Definición del Kernel 
*/
__global__ void sumar(int *a, int *b, int *r){
    *r = *a + *b;
}


int main(int argc, char** argv){

    int a = 11, b = 11, c = 0; // Copias del Host
    int *d_a,*d_b,*d_c; // Copias del Device

    //Reservo memoria en la GPU
    hipMalloc((void **)&d_a,sizeof(int));
    hipMalloc((void **)&d_b,sizeof(int));
    hipMalloc((void **)&d_c,sizeof(int));

    //Copio valores del Host al Device
    hipMemcpy(d_a,&a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,sizeof(int),hipMemcpyHostToDevice);

    //Lanzo el kernel

    sumar<<<1,1>>>(d_a,d_b,d_c);

    //Copio el resultado de Device a Host
    hipMemcpy(&c,d_c,sizeof(int),hipMemcpyDeviceToHost);

    cout << c << endl; 

    // Lipio variables de GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
